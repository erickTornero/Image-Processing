# include <hip/hip_runtime.h>

extern "C"
unsigned char * DFTimageCuda(unsigned char * data, int width, int height);

__global__ void processPixelVertical(unsigned char * data_dev, double * PkbReal_dev, double * PkbIm_dev, int width, int height){
    int posThread = blockIdx.x*blockDim.x + threadIdx.x;
    if(posThread < width*height){
        int k = posThread/width;
        int b = posThread - k*width;
        double sumReal = 0.0;
        double sumIm = 0.0;
        for(int a = 0; a < height; a++){
            double theta = -2.0*3.1416*k*a/height;
            sumReal += (double)data_dev[b + width*a]*cosf(theta);
            sumIm += (double)data_dev[b + width*a]*sinf(theta);
        }
        PkbReal_dev[b + width*k] = sumReal/(double)height;
        PkbIm_dev[b + width*k] = sumIm/(double)height; 
    }
}
__global__ void processPixelHorizontal(unsigned char *data_dev, double * PkbReal_dev, double * PkbIm_dev, int width, int height){
    int posThread = blockIdx.x*blockDim.x + threadIdx.x;
    if(posThread < width*height){
        int k = posThread/width;
        int l = posThread - k*width;
        double sumReal = 0.0;
        double sumIm = 0.0;
        for(int b = 0; b < width; b++){
            double theta = -2.0*3.1416*l*b/width;
            sumReal += (double)PkbReal_dev[b + k*width]*cosf(theta) - (double)PkbIm_dev[b+k*width]*sinf(theta);
            sumIm += (double)PkbReal_dev[b + k*width]*sinf(theta) + (double)PkbIm_dev[b+k*width]*cos(theta);
        }
        sumReal = sumReal/width;
        sumIm += sumIm/width;
        sumReal = sqrtf(sumReal*sumReal + sumIm*sumIm);
        data_dev[k*width + l] = (unsigned char) sumReal;
    }
}

unsigned char * DFTimageCuda(unsigned char * data, int width, int height){
    unsigned char * dataDev;
    hipMalloc((void**)&dataDev, width*height*sizeof(unsigned char));
    double * PkbRealDev;
    double * PkbImDev;
    hipMalloc((void**)&PkbRealDev, width*height*sizeof(double));
    hipMalloc((void**)&PkbImDev, width*height*sizeof(double));
    hipMemcpy(dataDev, data, width*height*sizeof(unsigned char), hipMemcpyHostToDevice);
    int nthreads = 1024;
    int nblocks = width*height/nthreads;
    if(width*height % nthreads > 0)
        nblocks++;
    processPixelVertical<<<nblocks, nthreads>>>(dataDev, PkbRealDev, PkbImDev, width, height);
    processPixelHorizontal<<<nblocks, nthreads>>>(dataDev, PkbRealDev, PkbImDev, width, height);
    unsigned char * Dft = new unsigned char[width*height];
    hipMemcpy(Dft, dataDev, width*height*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipFree(dataDev);
    hipFree(PkbRealDev);
    hipFree(PkbImDev);
    return Dft;
}
int main(){
    
}
